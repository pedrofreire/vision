#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

#include <iostream>


using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N)
{
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__device__ scalar_t deformable_im2col_bilinear(const scalar_t *bottom_data, const int data_width,
                                               const int height, const int width, scalar_t h, scalar_t w)
{

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(scalar_t argmax_h, scalar_t argmax_w,
                                        const int h, const int w, const int height, const int width)
{

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width)
  {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(scalar_t argmax_h, scalar_t argmax_w,
                                          const int height, const int width, const scalar_t *im_data,
                                          const int data_width, const int bp_dir)
{

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width)
  {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0)
  {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  }
  else if (bp_dir == 1)
  {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename scalar_t>
__global__ void deformable_im2col_gpu_kernel(const int n, const scalar_t *data_im, const scalar_t *data_offset,
                                             const int height, const int width, const int kernel_h, const int kernel_w,
                                             const int pad_h, const int pad_w, const int stride_h, const int stride_w,
                                             const int dilation_h, const int dilation_w, const int channel_per_deformable_group,
                                             const int batch_size, const int num_channels, const int deformable_group,
                                             const int height_col, const int width_col,
                                             scalar_t *data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    scalar_t *data_col_ptr = data_col + ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    //const scalar_t* data_im_ptr = data_im + ((b_col * num_channels + c_im) * height + h_in) * width + w_in;
    const scalar_t *data_im_ptr = data_im + (b_col * num_channels + c_im) * height * width;
    const scalar_t *data_offset_ptr = data_offset + (b_col * deformable_group + deformable_group_index) * 2 * kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i)
    {
      for (int j = 0; j < kernel_w; ++j)
      {
        const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
        const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
        const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
        scalar_t val = static_cast<scalar_t>(0);
        const scalar_t h_im = h_in + i * dilation_h + offset_h;
        const scalar_t w_im = w_in + j * dilation_w + offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
        {
          //const scalar_t map_h = i * dilation_h + offset_h;
          //const scalar_t map_w = j * dilation_w + offset_w;
          //const int cur_height = height - h_in;
          //const int cur_width = width - w_in;
          //val = deformable_im2col_bilinear(data_im_ptr, width, cur_height, cur_width, map_h, map_w);
          val = deformable_im2col_bilinear(data_im_ptr, width, height, width, h_im, w_im);
        }
        *data_col_ptr = val;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

void deformable_im2col(
    const at::Tensor data_im, const at::Tensor data_offset, const int channels,
    const int height, const int width, const int ksize_h, const int ksize_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int parallel_imgs,
    const int deformable_group, at::Tensor data_col)
{
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "deformable_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data<scalar_t>();
        scalar_t *data_col_ = data_col.data<scalar_t>();

        deformable_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
            num_kernels, data_im_, data_offset_, height, width, ksize_h, ksize_w,
            pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
            channel_per_deformable_group, parallel_imgs, channels, deformable_group,
            height_col, width_col, data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in deformable_im2col: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
__global__ void deformable_col2im_gpu_kernel(
    const int n, const scalar_t *data_col, const scalar_t *data_offset,
    const int channels, const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size, const int deformable_group,
    const int height_col, const int width_col,
    scalar_t *grad_im)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    const int j = (index / width_col / height_col / batch_size) % kernel_w;
    const int i = (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / batch_size / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int b = (index / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const scalar_t *data_offset_ptr = data_offset + (b * deformable_group + deformable_group_index) *
                                                        2 * kernel_h * kernel_w * height_col * width_col;
    const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
    const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
    const scalar_t cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const scalar_t cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const scalar_t cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++)
    {
      for (int dx = -2; dx <= 2; dx++)
      {
        if (cur_h + dy >= 0 && cur_h + dy < height &&
            cur_w + dx >= 0 && cur_w + dx < width &&
            abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1)
        {
          int cur_bottom_grad_pos = ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = get_gradient_weight(cur_inv_h_data, cur_inv_w_data, cur_h + dy, cur_w + dx, height, width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

void deformable_col2im(
    const at::Tensor data_col, const at::Tensor data_offset, const int channels,
    const int height, const int width, const int ksize_h,
    const int ksize_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int parallel_imgs, const int deformable_group,
    at::Tensor grad_im)
{

  // todo: make sure parallel_imgs is passed in correctly
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data<scalar_t>();
        scalar_t *grad_im_ = grad_im.data<scalar_t>();

        deformable_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
            num_kernels, data_col_, data_offset_, channels, height, width, ksize_h,
            ksize_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w, channel_per_deformable_group,
            parallel_imgs, deformable_group, height_col, width_col, grad_im_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
__global__ void deformable_col2im_coord_gpu_kernel(const int n, const scalar_t *data_col,
                                                   const scalar_t *data_im, const scalar_t *data_offset,
                                                   const int channels, const int height, const int width,
                                                   const int kernel_h, const int kernel_w,
                                                   const int pad_h, const int pad_w,
                                                   const int stride_h, const int stride_w,
                                                   const int dilation_h, const int dilation_w,
                                                   const int channel_per_deformable_group,
                                                   const int batch_size, const int offset_channels, const int deformable_group,
                                                   const int height_col, const int width_col, scalar_t *grad_offset)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    scalar_t val = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = (index / width_col / height_col) % offset_channels;
    int b = (index / width_col / height_col) / offset_channels;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const scalar_t *data_col_ptr = data_col + deformable_group_index * channel_per_deformable_group *
                                                  batch_size * width_col * height_col;
    const scalar_t *data_im_ptr = data_im + (b * deformable_group + deformable_group_index) *
                                                channel_per_deformable_group / kernel_h / kernel_w * height * width;
    const scalar_t *data_offset_ptr = data_offset + (b * deformable_group + deformable_group_index) * 2 *
                                                        kernel_h * kernel_w * height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group; col_c += col_step)
    {
      const int col_pos = (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i = (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr = (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr = (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out);
      const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
      const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
      scalar_t inv_h = h_in + i * dilation_h + offset_h;
      scalar_t inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width)
      {
        inv_h = inv_w = -2;
      }
      const scalar_t weight = get_coordinate_weight(
          inv_h, inv_w,
          height, width, data_im_ptr + cnt * height * width, width, bp_dir);
      val += weight * data_col_ptr[col_pos];
      cnt += 1;
    }

    grad_offset[index] = val;
  }
}

void deformable_col2im_coord(
    const at::Tensor data_col, const at::Tensor data_im, const at::Tensor data_offset,
    const int channels, const int height, const int width, const int ksize_h,
    const int ksize_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int parallel_imgs, const int deformable_group, at::Tensor grad_offset)
{

  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = height_col * width_col * 2 * ksize_h * ksize_w * deformable_group * parallel_imgs;
  int channel_per_deformable_group = channels * ksize_h * ksize_w / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_coord_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data<scalar_t>();
        const scalar_t *data_im_ = data_im.data<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data<scalar_t>();
        scalar_t *grad_offset_ = grad_offset.data<scalar_t>();

        deformable_col2im_coord_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
            num_kernels, data_col_, data_im_, data_offset_, channels, height, width,
            ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w, channel_per_deformable_group,
            parallel_imgs, 2 * ksize_h * ksize_w * deformable_group, deformable_group,
            height_col, width_col, grad_offset_);
      }));
}








void shape_check(at::Tensor input, at::Tensor offset, at::Tensor *gradOutput,
                 at::Tensor weight, int kH, int kW, int dH, int dW, int padH,
                 int padW, int dilationH, int dilationW, int group,
                 int deformable_group) {
  AT_CHECK(weight.ndimension() == 4,
           "4D weight tensor (nOutputPlane,nInputPlane,kH,kW) expected, "
           "but got: %s",
           weight.ndimension());

  AT_CHECK(weight.is_contiguous(), "weight tensor has to be contiguous");

  AT_CHECK(kW > 0 && kH > 0,
           "kernel size should be greater than zero, but got kH: %d kW: %d", kH,
           kW);

  AT_CHECK((weight.size(2) == kH && weight.size(3) == kW),
           "kernel size should be consistent with weight, ",
           "but got kH: %d kW: %d weight.size(2): %d, weight.size(3): %d", kH,
           kW, weight.size(2), weight.size(3));

  AT_CHECK(dW > 0 && dH > 0,
           "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);

  AT_CHECK(
      dilationW > 0 && dilationH > 0,
      "dilation should be greater than 0, but got dilationH: %d dilationW: %d",
      dilationH, dilationW);

  int ndim = input.ndimension();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  AT_CHECK(ndim == 3 || ndim == 4, "3D or 4D input tensor expected but got: %s",
           ndim);

  long nInputPlane = weight.size(1) * group;
  long inputHeight = input.size(dimh);
  long inputWidth = input.size(dimw);
  long nOutputPlane = weight.size(0);
  long outputHeight =
      (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  long outputWidth =
      (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;

  AT_CHECK(nInputPlane % deformable_group == 0,
           "input channels must divide deformable group size");

  if (outputWidth < 1 || outputHeight < 1)
    AT_ERROR(
        "Given input size: (%ld x %ld x %ld). "
        "Calculated output size: (%ld x %ld x %ld). Output size is too small",
        nInputPlane, inputHeight, inputWidth, nOutputPlane, outputHeight,
        outputWidth);

  AT_CHECK(input.size(1) == nInputPlane,
           "invalid number of input planes, expected: %d, but got: %d",
           nInputPlane, input.size(1));

  AT_CHECK((inputHeight >= kH && inputWidth >= kW),
           "input image is smaller than kernel");

  AT_CHECK((offset.size(2) == outputHeight && offset.size(3) == outputWidth),
           "invalid spatial size of offset, expected height: %d width: %d, but "
           "got height: %d width: %d",
           outputHeight, outputWidth, offset.size(2), offset.size(3));

  AT_CHECK((offset.size(1) == deformable_group * 2 * kH * kW),
           "invalid number of channels of offset");

  if (gradOutput != NULL) {
    AT_CHECK(gradOutput->size(dimf) == nOutputPlane,
             "invalid number of gradOutput planes, expected: %d, but got: %d",
             nOutputPlane, gradOutput->size(dimf));

    AT_CHECK((gradOutput->size(dimh) == outputHeight &&
              gradOutput->size(dimw) == outputWidth),
             "invalid size of gradOutput, expected height: %d width: %d , but "
             "got height: %d width: %d",
             outputHeight, outputWidth, gradOutput->size(dimh),
             gradOutput->size(dimw));
  }
}

int deform_conv_forward_cuda(at::Tensor input, at::Tensor weight,
                             at::Tensor offset, at::Tensor output,
                             at::Tensor columns, at::Tensor ones,
                             int dW, int dH, int padW, int padH,
                             int dilationW, int dilationH, int group,
                             int deformable_group, int im2col_step) {
  // todo: resize columns to include im2col: done
  // todo: add im2col_step as input
  // todo: add new output buffer and transpose it to output (or directly
  // transpose output) todo: possibly change data indexing because of
  // parallel_imgs

  int kH = weight.size(3);
  int kW = weight.size(2);

  shape_check(input, offset, NULL, weight, kH, kW, dH, dW, padH, padW,
              dilationH, dilationW, group, deformable_group);
  at::DeviceGuard guard(input.device());
  
  input = input.contiguous();
  offset = offset.contiguous();
  weight = weight.contiguous();

  // todo: assert batchsize dividable by im2col_step

  long batchSize = input.size(0);
  long nInputPlane = input.size(1);
  long inputHeight = input.size(2);
  long inputWidth = input.size(3);

  long nOutputPlane = weight.size(0);

  long outputWidth =
      (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight =
      (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  AT_CHECK((offset.size(0) == batchSize), "invalid batch size of offset");

  output = output.view({batchSize / im2col_step, im2col_step, nOutputPlane,
                        outputHeight, outputWidth});
  columns = at::zeros(
      {nInputPlane * kW * kH, im2col_step * outputHeight * outputWidth},
      input.options());

  if (ones.ndimension() != 2 ||
      ones.size(0) * ones.size(1) < outputHeight * outputWidth) {
    ones = at::ones({outputHeight, outputWidth}, input.options());
  }

  input = input.view({batchSize / im2col_step, im2col_step, nInputPlane,
                      inputHeight, inputWidth});
  offset =
      offset.view({batchSize / im2col_step, im2col_step,
                   deformable_group * 2 * kH * kW, outputHeight, outputWidth});

  at::Tensor output_buffer =
      at::zeros({batchSize / im2col_step, nOutputPlane,
                 im2col_step * outputHeight, outputWidth},
                output.options());

  output_buffer = output_buffer.view(
      {output_buffer.size(0), group, output_buffer.size(1) / group,
       output_buffer.size(2), output_buffer.size(3)});

  for (int elt = 0; elt < batchSize / im2col_step; elt++) {
    deformable_im2col(input[elt], offset[elt], nInputPlane, inputHeight,
                      inputWidth, kH, kW, padH, padW, dH, dW, dilationH,
                      dilationW, im2col_step, deformable_group, columns);

    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    weight = weight.view({group, weight.size(0) / group, weight.size(1),
                          weight.size(2), weight.size(3)});

    // std::cout << "columns:\n";
    // std::cout << columns << "\n\n";

    for (int g = 0; g < group; g++) {
      output_buffer[elt][g] = output_buffer[elt][g]
                                  .flatten(1)
                                  .addmm_(weight[g].flatten(1), columns[g])
                                  .view_as(output_buffer[elt][g]);
    }
  }

  output_buffer = output_buffer.view(
      {output_buffer.size(0), output_buffer.size(1) * output_buffer.size(2),
       output_buffer.size(3), output_buffer.size(4)});

  output_buffer = output_buffer.view({batchSize / im2col_step, nOutputPlane,
                                      im2col_step, outputHeight, outputWidth});
  output_buffer.transpose_(1, 2);
  output.copy_(output_buffer);
  output = output.view({batchSize, nOutputPlane, outputHeight, outputWidth});

  // std::cout << "output:\n";
  // std::cout << output << "\n\n";

  input = input.view({batchSize, nInputPlane, inputHeight, inputWidth});
  offset = offset.view(
      {batchSize, deformable_group * 2 * kH * kW, outputHeight, outputWidth});

  return 1;
}






/*

template <typename T>
__global__ void DCNForward(
    const T* input,
    const T* offset,
    const T* weight,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int deformable_groups,
    const int im2col_step,
    T* output) {
  output[0] = input[0] * input[0];
}
*/
//*

at::Tensor DCN_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& offset,
    const at::Tensor& weight,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int deformable_groups,
    const int im2col_step) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto n_channels = weight.size(0);
  auto in_size = input.size(2);
  auto kernel_size = dilation * (weight.size(2) - 1) + 1;
  auto out_size = (in_size + (2 * padding) - kernel_size) / stride + 1;

  at::Tensor output = at::zeros({batch_size, n_channels, out_size, out_size}, input.options());

  at::Tensor buf0 = at::zeros({1}, input.options());
  at::Tensor buf1 = at::zeros({1}, input.options());

  int in_size0 = input.size(0);
  auto cur_im2col_step = std::min(in_size0, im2col_step);
  TORCH_CHECK(in_size0 % cur_im2col_step == 0);

  deform_conv_forward_cuda(
      input, weight, offset, output, buf0, buf1,
      stride, stride,
      padding, padding,
      dilation, dilation,
      groups, deformable_groups,
      cur_im2col_step);

  /*
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "DCN_forward", [&] {
    DCNForward<scalar_t><<<1, 1, 0, stream>>>(
        input.contiguous().data_ptr<scalar_t>(),
        weight.contiguous().data_ptr<scalar_t>(),
        offset.contiguous().data_ptr<scalar_t>(),
        buf0.contiguous().data_ptr<scalar_t>(),
        buf1.contiguous().data_ptr<scalar_t>(),
        stride,
        stride,
        padding,
        padding,
        dilation,
        dilation,
        groups,
        deformable_groups,
        im2col_step,
        output.data_ptr<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  */
  return output;
}
// */



template <typename T>
__global__ void DCNBackward(
    const T* grad_output,
    const T* input,
    T* grad_input) {
  grad_input[0] = 2 * input[0] * grad_output[0];
}

at::Tensor DCN_backward_cuda(
    const at::Tensor& grad, const at::Tensor& input) {
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(grad.device());

  at::Tensor grad_input =
      at::zeros({1}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "DCN_backward", [&] {
    DCNBackward<scalar_t><<<1, 1, 0, stream>>>(
        grad.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        grad_input.data_ptr<scalar_t>());
  });
  return grad_input;
}


