#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

void deform_conv_forward_cuda(at::Tensor input, at::Tensor weight,
                             at::Tensor offset, at::Tensor output,
                             at::Tensor columns, at::Tensor ones,
                             int dW, int dH, int padW, int padH,
                             int dilationW, int dilationH, int group,
                             int deformable_group, int im2col_step) {
}






/*

template <typename T>
__global__ void DCNForward(
    const T* input,
    const T* offset,
    const T* weight,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int deformable_groups,
    const int im2col_step,
    T* output) {
  output[0] = input[0] * input[0];
}
*/
//*

at::Tensor DCN_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& offset,
    const at::Tensor& weight,
    const int stride,
    const int padding,
    const int dilation,
    const int groups,
    const int deformable_groups,
    const int im2col_step) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");

  at::cuda::CUDAGuard device_guard(input.device());

  auto batch_size = input.size(0);
  auto n_channels = weight.size(0);
  auto in_size = input.size(2);
  auto kernel_size = dilation * (weight.size(2) - 1) - 1;
  auto out_size = (in_size + (2 * padding) - kernel_size) / stride + 1;

  at::Tensor output = at::zeros({batch_size, n_channels, out_size, out_size}, input.options());

  at::Tensor buf0 = at::zeros({1}, input.options());
  at::Tensor buf1 = at::zeros({1}, input.options());

  int in_size0 = input.size(0);
  auto cur_im2col_step = std::min(in_size0, im2col_step);
  TORCH_CHECK(in_size0 % cur_im2col_step == 0);

  deform_conv_forward_cuda(
      input, weight, offset, output, buf0, buf1,
      stride, stride,
      padding, padding,
      dilation, dilation,
      groups, deformable_groups,
      cur_im2col_step);

  /*
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "DCN_forward", [&] {
    DCNForward<scalar_t><<<1, 1, 0, stream>>>(
        input.contiguous().data_ptr<scalar_t>(),
        weight.contiguous().data_ptr<scalar_t>(),
        offset.contiguous().data_ptr<scalar_t>(),
        buf0.contiguous().data_ptr<scalar_t>(),
        buf1.contiguous().data_ptr<scalar_t>(),
        stride,
        stride,
        padding,
        padding,
        dilation,
        dilation,
        groups,
        deformable_groups,
        im2col_step,
        output.data_ptr<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  */
  return output;
}
// */



template <typename T>
__global__ void DCNBackward(
    const T* grad_output,
    const T* input,
    T* grad_input) {
  grad_input[0] = 2 * input[0] * grad_output[0];
}

at::Tensor DCN_backward_cuda(
    const at::Tensor& grad, const at::Tensor& input) {
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(grad.device());

  at::Tensor grad_input =
      at::zeros({1}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "DCN_backward", [&] {
    DCNBackward<scalar_t><<<1, 1, 0, stream>>>(
        grad.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        grad_input.data_ptr<scalar_t>());
  });
  return grad_input;
}


