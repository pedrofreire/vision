#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

template <typename T>
__global__ void RoIPoolForward(
    const T* input,
    T* output) {
  output[0] = 10;
}

template <typename T>
__global__ void RoIPoolBackward(
    const T* grad_output,
    T* grad_input) {
  grad_input[0] = 20;
}

at::Tensor ROIPool_forward_cuda(
    const at::Tensor& input) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");

  at::cuda::CUDAGuard device_guard(input.device());

  at::Tensor output = at::zeros(
      {1}, input.options());

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "ROIPool_forward", [&] {
    RoIPoolForward<scalar_t><<<grid, block, 0, stream>>>(
        input.contiguous().data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  return output;
}

at::Tensor ROIPool_backward_cuda(
    const at::Tensor& grad) {
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(grad.device());

  at::Tensor grad_input =
      at::zeros({1}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "ROIPool_backward", [&] {
    RoIPoolBackward<scalar_t><<<grid, block, 0, stream>>>(
        grad.data_ptr<scalar_t>(),
        grad_input.data_ptr<scalar_t>());
  });
  return grad_input;
}
